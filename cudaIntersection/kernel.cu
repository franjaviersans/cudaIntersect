#include "hip/hip_runtime.h"
#include "kernel.cuh"


//Ray-Triangle function to calculate the intersection
__device__ bool ray_triangle( const float3 V1,  // Triangle vertices
                           const float3 V2,
                           const float3 V3,
                           const float3 O,  //Ray origin
                           const float3 D  //Ray direction
						   )
{

	#ifdef ALLFALSE
		return false;
	#endif

	#ifdef ALLTRUE
		return true;
	#endif

	float3 e1, e2;  //Edge1, Edge2
	float3 P, Q, T;
	float det, inv_det, u, v;
	float t;
 
	//Find vectors for two edges sharing V1
	SUB(e1, V2, V1);
	SUB(e2, V3, V1);
	//Begin calculating determinant - also used to calculate u parameter
	CROSS(P, D, e2);
	//if determinant is near zero, ray lies in plane of triangle
	det = DOT(e1, P);
	//NOT CULLING
	if(det > -EPSILON && det < EPSILON) return false;
	inv_det = 1.f / det;
 
	//calculate distance from V1 to ray origin
	SUB(T, O, V1);
 
	//Calculate u parameter and test bound
	u = DOT(T, P) * inv_det;
	//The intersection lies outside of the triangle
	if(u < 0.f || u > 1.f) return false;
 
	//Prepare to test v parameter
	CROSS(Q, T, e1);
 
	//Calculate V parameter and test bound
	v = DOT(D, Q) * inv_det;
	//The intersection lies outside of the triangle
	if(v < 0.f || u + v  > 1.f) return false;
 
	t = DOT(e2, Q) * inv_det;
 
	if(t > EPSILON && t < 1.0f) { //ray intersection
		//We only consider the [0.0,1.0] interval because is the distance between the origin and the point in the surface C
		return true;
	}
 
	// No hit, no win
	return false;
}

//A simple geometry test to be done in the points of A
//For now this is only a dommy test
__device__ bool Test2(float3 v, unsigned int i){

	#ifdef ALLFALSE
		return false;
	#endif

	#ifdef ALLTRUE
		return true;
	#endif

	//return v.x * v.x + v.y * v.y < v.z * v.z + i * i;
	return false;
}

__global__ void Intercept(const float3 * const p1, const float3 * const p2,
			   const float3 * const A, const uint3 * const B,
			   const unsigned int sizeC, const unsigned int sizeA,
			   const unsigned int sizeB, 
			   const float * const x,
			   bool * const globalinter)
{

	//Shared memory declaration
	extern __shared__ char buffer[];

	//All point of C will be stored in the array dir in shared memory
	float3 * dir = (float3 *)&buffer[0];
    float3 * origin = (float3 *)&buffer[sizeC * sizeof(float3)]; //The origin will be shared to
	float * lt = (float *)&buffer[sizeC * sizeof(float3) + sizeof(float3)]; //The shared transformation matrix
	bool * sharedinter = (bool *)&buffer[sizeC * sizeof(float3) + sizeof(float3) + 16 *sizeof(float)]; //A boolean to know if the test has to stop

	//Id of the thread within a block and within the grid
	unsigned int tid = threadIdx.x;
	unsigned int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

	//Auxiliar variables
	float3 v0, v1, v2, vaux;
	bool inter = false;
	
	//if it is the first thread of the block
	if(tid == 0)
	{
		//Set global boolean to false (there has been no ray-triangle intersection
		*sharedinter = false;
		vaux = *p1; //Copy the data of the origin of the ray
		MULT((*origin), x, vaux); //Transfor the point. This is the only transformation done with global transformation data!!!
	}
	if(tid < 16) lt[tid] = x[tid]; //16 values of the 4x4 transformation matrix
	__syncthreads(); //Wait to all the threads in the block

	//Copy all the data of C
	if(tid < sizeC)
	{
		//Copy a point of C to local data
		v0 = p2[tid];

		//Transform the point 
		MULT(vaux, lt, v0);

		//store the direction of the ray in shared memory x(p2) - x(p1)
		dir[tid].x = vaux.x - (*origin).x;
		dir[tid].y = vaux.y - (*origin).y;
		dir[tid].z = vaux.z - (*origin).z;
	}
	__syncthreads(); //Wait to all the threads in the block

	
	//Each thread works with one triangle in the surface (A, B)
	if(globalTid < sizeB){
		uint3 id = B[globalTid];

		//Store the points of the triangles in local memory
		v0 = A[id.x]; //Point 0
		v1 = A[id.y]; //Point 1
		v2 = A[id.z]; //Point 2

		//First test. Ray-Triangle Intersection
		unsigned int i;
		for(i=0; i < sizeC && !*globalinter;++i)  //For all the points in C do the intersection test
		{
			inter = ray_triangle(v0, v1, v2, (*origin), dir[i]); //Intersection function with the 3 points of the triangle, the origin, and the ith direction
			#ifdef ALLTEST
				if(inter) *globalinter = false;
			#else
				if(inter) *globalinter = true; //If and intersection has ocurred, set the global intersection flag to true
			#endif
		}
	}

}

//Kernel to do the second test
//Test in every point of the surface (A, B)
__global__ void SecondTest(	const float3 * const A, const unsigned int sizeA, bool * const globalinter)
{
	//Id of the thread in the grid
	unsigned int globalTid = blockDim.x * blockIdx.x + threadIdx.x;

	bool inter = false;

	float3 v;
	
	if(globalTid < sizeA)
	{
		v = A[globalTid]; //Copy the data of the point into local memory

		//Second Test
		unsigned int j;
		for(j=0;j < N && !*globalinter;++j) //Do the tests
		{
			inter = Test2(v, j);
			#ifdef ALLTEST
				if(inter) *globalinter = false;
			#else
				if(inter) *globalinter = true;
			#endif
		}
		//if(globalTid == 0)printf("%d \n",j);
	}
}

bool CUDA::CudaIntercept(float &time, vector<Transformation> *vTrans){
	bool h_inter = false;

	Transformation t;

	//Generate a random transform with scaling, translating and rotating
	t.m_fScalar = (rand() % RAND_MAX) / float(RAND_MAX);
	t.m_fTranslationx = (rand() % RAND_MAX) / float(RAND_MAX/2.0f) -1.0f;
	t.m_fTranslationy = (rand() % RAND_MAX) / float(RAND_MAX/2.0f) -1.0f;
	t.m_fTranslationz = (rand() % RAND_MAX) / float(RAND_MAX/2.0f) -1.0f;

	glm::vec3 rotation_angle = glm::normalize(glm::vec3((rand() % RAND_MAX) / float(RAND_MAX/2.0f) -1.0f, (rand() % RAND_MAX) / float(RAND_MAX/2.0f) -1.0f, (rand() % RAND_MAX) / float(RAND_MAX/2.0f) -1.0f));
	t.m_fRotationAngle = (rand() % RAND_MAX) / float(RAND_MAX/2.0f) -1.0f;
	t.m_fRotationVectorx = rotation_angle.x;
	t.m_fRotationVectory = rotation_angle.y;
	t.m_fRotationVectorz = rotation_angle.z;

	vTrans->push_back(t);

	//Generate quaternion
	glm::quat quater = glm::quat(t.m_fRotationAngle, glm::normalize(glm::vec3(rotation_angle)));

	//Construct the transformation matrix with glm
	glm::mat4 RotationMat = glm::mat4_cast(glm::normalize(quater));
	glm::mat4 mCTransfor =	glm::translate(glm::mat4(), glm::vec3( t.m_fTranslationx , t.m_fTranslationy, t.m_fTranslationz)) * 
							RotationMat * 
							glm::scale(glm::mat4(), glm::vec3(t.m_fScalar )) * 
							glm::mat4();

	
	

	//Copy the information of the transform an the global itersection boolean
	checkCudaErrors(hipMemcpy(d_x, glm::value_ptr(mCTransfor),  16 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_inter, &h_inter,  sizeof(bool), hipMemcpyHostToDevice));

	//Each thread for each triangle
	dim3 BlockDim(128, 1, 1); //128 threads per block
	dim3 GridDim((sizeB + BlockDim.x)/BlockDim.x, 1, 1); 

	//First test with timer
	GpuTimer timer;
	timer.Start();
	Intercept<<< GridDim, BlockDim, sizeC * sizeof(float3) + sizeof(float3) + sizeof(bool) + 16 *sizeof(float) >>>(d_p1, d_p2, d_A, d_B, sizeC, sizeA, sizeB, d_x, d_inter);
	timer.Stop();

	time += timer.Elapsed();
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError()); //Check for errors

	checkCudaErrors(hipMemcpy(&h_inter, d_inter,  sizeof(bool), hipMemcpyDeviceToHost)); //Copy the GPU global intersection variable to know if there was an intersection
	
	if(!h_inter) //if there has no been an intersection then do the secon test
	{

		//One thread for each point in the surface
		dim3 BlockDim2(128, 1, 1); //128 threads per block
		dim3 GridDim2((sizeA + BlockDim.x)/BlockDim.x, 1, 1);

		//Second test with timer
		timer.Start();
		SecondTest<<< GridDim2, BlockDim2>>>(d_A, sizeA, d_inter);
		timer.Stop();

		time += timer.Elapsed();
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());//Check for errors

		checkCudaErrors(hipMemcpy(&h_inter, d_inter,  sizeof(bool), hipMemcpyDeviceToHost));  //Copy the GPU global intersection variable to know if there was an intersection
	
	}

	return h_inter;
}

//Function to copy all the geometry information into the GPU
__host__ void CUDA::Init(float3 * A, uint3  * B, float3 *C, unsigned int sA, unsigned int sB, unsigned int sC){

	//Check if object C can be stored in shared memory
	hipDeviceProp_t prop;

	checkCudaErrors(hipSetDevice(0));

	checkCudaErrors( hipGetDeviceProperties( &prop, 0 ) );

	if(prop.sharedMemPerBlock < sC * sizeof(float3))
	{
		printf("Surface C cannot be stored in shared memory. Other approach should be use\n");
		exit(0);
	}

	sizeA = sA;
	sizeB = sB;
	sizeC = sC;

	/* initialize random seed: */
	srand (unsigned int(time(NULL)));

	

	//Allocate memory on the GPU

	checkCudaErrors(hipMalloc((void**)&d_p1, sizeof(float3)));
	checkCudaErrors(hipMalloc((void**)&d_p2, sizeC * sizeof(float3)));
	checkCudaErrors(hipMalloc((void**)&d_A, sizeA * sizeof(float3)));
	checkCudaErrors(hipMalloc((void**)&d_B, sizeB * sizeof(uint3)));
	checkCudaErrors(hipMalloc((void**)&d_x, 16 * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_inter, sizeof(bool)));
	
	//Send information to the GPU
	checkCudaErrors(hipMemcpy(d_p1, C, sizeof(float3), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_p2, C, sizeC * sizeof(float3), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_A, A, sizeA * sizeof(float3), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_B, B, sizeB * sizeof(uint3), hipMemcpyHostToDevice));
}

//Function to free memory
__host__ void CUDA::Destroy(){
	//Free memory
	checkCudaErrors(hipFree(d_p1));
	checkCudaErrors(hipFree(d_p2));
	checkCudaErrors(hipFree(d_A));
	checkCudaErrors(hipFree(d_B));
	checkCudaErrors(hipFree(d_x));
	checkCudaErrors(hipFree(d_inter));
}